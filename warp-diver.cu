
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void divergenceExample(int *array) {
    int tid = threadIdx.x;
    if (tid % 2 == 0) {
        // Even threads perform this operation
        array[tid] = tid * 2;
    } else {
        // Odd threads perform this operation
        array[tid] = tid * 3;
    }
}

int main() {
    const int arraySize = 16;
    int *d_array;
    hipMalloc(&d_array, arraySize * sizeof(int));

    // Launch the kernel with 1 block and 16 threads
    divergenceExample<<<1, arraySize>>>(d_array);

    // Copy the result back to host
    int h_array[arraySize];
    hipMemcpy(h_array, d_array, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_array);

    return 0;
}
